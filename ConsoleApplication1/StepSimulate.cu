#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "PARTICLE_SYSTEM.h"
#include <stdio.h>
#include <stdlib.h>
#define THREAD 1024
#define attractFactor  0.070f / h//0.045f / h;
#define velocityFactor  0.005f / h;
#define pa d_data[idx][i * THREAD + pidx]
#define pb d_data[nidx][j]
#pragma unroll
#define BOXSIZE 0.4
//#define dt 0.01


typedef struct vec3d {
	float x;
	float y;
	float z;
}vec3d;
__device__ vec3d newVec3d(float x, float y, float z) {
	vec3d temp;
	temp.x = x; temp.y = y; temp.z = z;
	return temp;
}
typedef struct particle {
	vec3d position;
	vec3d velocity;
	vec3d force;
	vec3d acceleration;
	//vec3d normal;
	float density;
	float scaleFactor;
	float pressure;
	bool flag;
	bool exSPHflag;
	//bool exSPHbottomflag;
	//bool exSPHsurfaceflag;
	//float exdensity;
	//float expressure;
	//vec3d exacceleration;
	int id;
	int sumid;
}particle;
typedef struct wall {
	vec3d normal;
	vec3d point;
}wall;
//取出grid(x, y, z)中的粒子

inline __host__ __device__ int grid_index(int x, int y, int z, int xRes, int yRes, int zRes) {
	return x + y * xRes + z * xRes*yRes; // return max = xRes*yRes*zRes - 1
}
//核函数
__device__ float Wpoly6(float radiusSquared)
{
	const float coefficient = 315.0f / (64.0f*M_PI*pow(h, 9));
	const float hSquared = h * h;
	return coefficient * pow(hSquared - radiusSquared, 3);
}
__device__ void Wpoly6Gradient(vec3d& diffPosition, float radiusSquared, vec3d& gradient)
{
	const float coefficient = -945.0f / (32.0f*M_PI*pow(h, 9));
	const float hSquared = h * h;
	//gradient = coefficient * pow(hSquared - radiusSquared, 2) * diffPosition;
	gradient.x = coefficient * pow(hSquared - radiusSquared, 2) * diffPosition.x;
	gradient.y = coefficient * pow(hSquared - radiusSquared, 2) * diffPosition.y;
	gradient.z = coefficient * pow(hSquared - radiusSquared, 2) * diffPosition.z;
}
__device__ float Wpoly6Laplacian(float radiusSquared)
{
	const float coefficient = -945.0f / (32.0f*M_PI*pow(h, 9));
	const float hSquared = h * h;
	return coefficient * (hSquared - radiusSquared) * (3.0*hSquared - 7.0*radiusSquared);
}
__device__ void WspikyGradient(vec3d& diffPosition, float radiusSquared, vec3d& gradient)
{
	const float coefficient = -45.0f / (M_PI*pow(h, 6));
	float radius = sqrt(radiusSquared);
	gradient.x = coefficient * pow(h - radius, 2) * diffPosition.x / radius;
	gradient.y = coefficient * pow(h - radius, 2) * diffPosition.y / radius;
	gradient.z = coefficient * pow(h - radius, 2) * diffPosition.z / radius;
}
__device__ float WviscosityLaplacian(float radiusSquared)
{
	const float coefficient = 45.0f / (M_PI*pow(h, 6));
	float radius = sqrt(radiusSquared);
	return coefficient * (h - radius);
}
__device__ void collisionForce_z(particle &p, wall *d_wall, int wSize) {
	int i = 0;
	for (i = 0; i < wSize; i++)
	{
		wall wall = d_wall[i];
		float d = (wall.point.x - p.position.x)*wall.normal.x +
			(wall.point.y - p.position.y)*wall.normal.y +
			(wall.point.z - p.position.z)*wall.normal.z + 0.015f; // d为穿透深度
		if (d > 0.0f)
		{
			p.acceleration.x += WALL_K * wall.normal.x * d;
			p.acceleration.y += WALL_K * wall.normal.y * d;
			p.acceleration.z += WALL_K * wall.normal.z * d;
			p.acceleration.x += (WALL_DAMPING * p.velocity.x *wall.normal.x +
				WALL_DAMPING * p.velocity.y *wall.normal.y +
				WALL_DAMPING * p.velocity.z *wall.normal.z) * wall.normal.x;
			p.acceleration.y += (WALL_DAMPING * p.velocity.x *wall.normal.x +
				WALL_DAMPING * p.velocity.y *wall.normal.y +
				WALL_DAMPING * p.velocity.z *wall.normal.z) * wall.normal.y;
			p.acceleration.z += (WALL_DAMPING * p.velocity.x *wall.normal.x +
				WALL_DAMPING * p.velocity.y *wall.normal.y +
				WALL_DAMPING * p.velocity.z *wall.normal.z) * wall.normal.z;
		}
	}
}
int xRes, yRes, zRes;
particle **h_data;   // host field
particle **d_data;   //device field
particle **new_data; //更新后的网格
wall *h_wall, *d_wall;
int *h_size, *d_size;
int *new_size; //更新后的size
inline void CHECK(hipError_t err)//错误处理函数
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error: %s.\n", hipGetErrorString(err));
		return;
	}
}
//参数：d_data设备内存上的粒子，分为若干个grid，d_size
__global__ void caculateDensity(particle** d_data, int* d_size, int xRes, int yRes, int zRes) {
	int idx = grid_index((int)blockIdx.x, (int)blockIdx.y, (int)blockIdx.z, xRes, yRes, zRes);
	// 访问grid(blockIdx.x, blockIdx.y, blockIdx.z)
	int pidx = (int)threadIdx.x;
	int i;
	if (idx < xRes*yRes*zRes) {
		for (i = 0; i * THREAD + pidx < d_size[idx]; ++i) {
			if (d_data[idx][i * THREAD + pidx].exSPHflag) { //对于水体粒子求密度
				d_data[idx][i * THREAD + pidx].density = 0;
				//caculate density
				int x, y, z;
				int j;
				int nsize, nidx;
				for (x = (int)blockIdx.x - 1; x <= (int)blockIdx.x + 1; ++x) {
					if (x < 0) continue;
					if (x >= xRes) break;
					for (y = (int)blockIdx.y - 1; y <= (int)blockIdx.y + 1; ++y) {
						if (y < 0) continue;
						if (y >= yRes) break;
						for (z = (int)blockIdx.z - 1; z <= (int)blockIdx.z + 1; ++z) {
							if (z < 0) continue;
							if (z >= zRes) break;
							nidx = grid_index(x, y, z, xRes, yRes, zRes);
							if (nidx < xRes*yRes*zRes) {
								nsize = d_size[nidx];
								for (j = 0; j < nsize; ++j) {
									if (d_data[nidx][j].exSPHflag) {	//周围的水体粒子才对密度值有贡献，牵引粒子不影响密度
										vec3d diffPosition;
										diffPosition.x = d_data[idx][i * THREAD + pidx].position.x - d_data[nidx][j].position.x;
										diffPosition.y = d_data[idx][i * THREAD + pidx].position.y - d_data[nidx][j].position.y;
										diffPosition.z = d_data[idx][i * THREAD + pidx].position.z - d_data[nidx][j].position.z;
										float radiusSquared = diffPosition.x * diffPosition.x + diffPosition.y * diffPosition.y + diffPosition.z * diffPosition.z;
										if (radiusSquared <= h * h)
											d_data[idx][i * THREAD + pidx].density += Wpoly6(radiusSquared);
									}
								}
							}
						}
					}
				}
				d_data[idx][i * THREAD + pidx].density *= PARTICLE_MASS;
				d_data[idx][i * THREAD + pidx].pressure = GAS_STIFFNESS * (d_data[idx][i * THREAD + pidx].density - REST_DENSITY);
			}
		}
	}
}
__global__ void caculateScaleFactor(particle** d_data, int* d_size, int xRes, int yRes, int zRes) {
	int idx = grid_index((int)blockIdx.x, (int)blockIdx.y, (int)blockIdx.z, xRes, yRes, zRes);
	// 访问grid(blockIdx.x, blockIdx.y, blockIdx.z)
	int pidx = (int)threadIdx.x;
	int i;
	if (idx < xRes*yRes*zRes) {
		for (i = 0; i * THREAD + pidx < d_size[idx]; ++i) {
			if (!d_data[idx][i * THREAD + pidx].exSPHflag) {	//对于牵引粒子计算牵引力系数
				d_data[idx][i * THREAD + pidx].scaleFactor = 0;
				//caculate density
				int x, y, z;
				int j;
				int nsize, nidx;
				for (x = (int)blockIdx.x - 1; x <= (int)blockIdx.x + 1; ++x) {
					if (x < 0) continue;
					if (x >= xRes) break;
					for (y = (int)blockIdx.y - 1; y <= (int)blockIdx.y + 1; ++y) {
						if (y < 0) continue;
						if (y >= yRes) break;
						for (z = (int)blockIdx.z - 1; z <= (int)blockIdx.z + 1; ++z) {
							if (z < 0) continue;
							if (z >= zRes) break;
							nidx = grid_index(x, y, z, xRes, yRes, zRes);
							if (nidx < xRes*yRes*zRes) {
								nsize = d_size[nidx];
								for (j = 0; j < nsize; ++j) {
									if (d_data[nidx][j].exSPHflag) {	
										vec3d diffPosition;
										diffPosition.x = d_data[idx][i * THREAD + pidx].position.x - d_data[nidx][j].position.x;
										diffPosition.y = d_data[idx][i * THREAD + pidx].position.y - d_data[nidx][j].position.y;
										diffPosition.z = d_data[idx][i * THREAD + pidx].position.z - d_data[nidx][j].position.z;
										float radiusSquared = diffPosition.x * diffPosition.x + diffPosition.y * diffPosition.y + diffPosition.z * diffPosition.z;
										if (radiusSquared <= h * h)
											d_data[idx][i * THREAD + pidx].scaleFactor += 1.0 / d_data[nidx][j].density * Wpoly6(radiusSquared);
									}
								}
							}
						}
					}
				}
				d_data[idx][i * THREAD + pidx].scaleFactor *= PARTICLE_MASS;
				d_data[idx][i * THREAD + pidx].scaleFactor = (1.0 < (d_data[idx][i * THREAD + pidx].scaleFactor)) ? 0.0 : (1.0 - d_data[idx][i * THREAD + pidx].scaleFactor);
			}
		}
	}
}
__global__ void caculateAllForce(particle** d_data, int* d_size, int xRes, int yRes, int zRes, int iter) {
	vec3d f_pressure,
		f_viscosity;
	vec3d	f_gravity, f_surface,
		colorFieldNormal,
		f_attract,
		f_velocity;
	float sqrtc;
	int idx = grid_index((int)blockIdx.x, (int)blockIdx.y, (int)blockIdx.z, xRes, yRes, zRes);
	// 访问grid(blockIdx.x, blockIdx.y, blockIdx.z)
	int pidx = (int)threadIdx.x;
	int i;
	if (idx < xRes*yRes*zRes) {
		for (i = 0; i * THREAD + pidx < d_size[idx]; ++i) {
			if (d_data[idx][i * THREAD + pidx].exSPHflag) {		//对于水体粒子，计算其受力情况
				f_pressure = newVec3d(0, 0, 0),
					f_viscosity = newVec3d(0, 0, 0),
					f_surface = newVec3d(0, 0, 0),
					f_gravity = newVec3d(0.0, GRAVITY_ACCELERATION * d_data[idx][i * THREAD + pidx].density, 0.0),
					colorFieldNormal = newVec3d(0, 0, 0),
					f_attract = newVec3d(0, 0, 0),
					f_velocity = newVec3d(0, 0, 0);
				float colorFieldLaplacian = 0.0;
				//caculate density
				int x, y, z;
				int j;
				int nsize, nidx;
				for (x = (int)blockIdx.x - 1; x <= (int)blockIdx.x + 1; ++x) {
					if (x < 0) continue;
					if (x >= xRes) break;
					for (y = (int)blockIdx.y - 1; y <= (int)blockIdx.y + 1; ++y) {
						if (y < 0) continue;
						if (y >= yRes) break;
						for (z = (int)blockIdx.z - 1; z <= (int)blockIdx.z + 1; ++z) {
							if (z < 0) continue;
							if (z >= zRes) break;
							nidx = grid_index(x, y, z, xRes, yRes, zRes);
							if (nidx < xRes*yRes*zRes) {
								for (j = 0; j < d_size[nidx]; ++j) {
									vec3d diffPosition;
									diffPosition.x = d_data[idx][i * THREAD + pidx].position.x - d_data[nidx][j].position.x;
									diffPosition.y = d_data[idx][i * THREAD + pidx].position.y - d_data[nidx][j].position.y;
									diffPosition.z = d_data[idx][i * THREAD + pidx].position.z - d_data[nidx][j].position.z;
									float radiusSquared = diffPosition.x * diffPosition.x + diffPosition.y * diffPosition.y + diffPosition.z * diffPosition.z;
									if (radiusSquared <= h * h && d_data[nidx][j].exSPHflag) { //水体粒子提供的压力和粘性力
										vec3d poly6Gradient, spikyGradient;
										Wpoly6Gradient(diffPosition, radiusSquared, poly6Gradient);
										WspikyGradient(diffPosition, radiusSquared, spikyGradient);
										if (d_data[idx][i * THREAD + pidx].id != d_data[nidx][j].id) {
											f_pressure.x += (d_data[idx][i * THREAD + pidx].pressure / pow(d_data[idx][i * THREAD + pidx].density, 2) + d_data[nidx][j].pressure / pow(d_data[nidx][j].density, 2))*spikyGradient.x;
											f_pressure.y += (d_data[idx][i * THREAD + pidx].pressure / pow(d_data[idx][i * THREAD + pidx].density, 2) + d_data[nidx][j].pressure / pow(d_data[nidx][j].density, 2))*spikyGradient.y;
											f_pressure.z += (d_data[idx][i * THREAD + pidx].pressure / pow(d_data[idx][i * THREAD + pidx].density, 2) + d_data[nidx][j].pressure / pow(d_data[nidx][j].density, 2))*spikyGradient.z;
											f_viscosity.x += (d_data[nidx][j].velocity.x - d_data[idx][i * THREAD + pidx].velocity.x) * WviscosityLaplacian(radiusSquared) / d_data[nidx][j].density;
											f_viscosity.y += (d_data[nidx][j].velocity.y - d_data[idx][i * THREAD + pidx].velocity.y) * WviscosityLaplacian(radiusSquared) / d_data[nidx][j].density;
											f_viscosity.z += (d_data[nidx][j].velocity.z - d_data[idx][i * THREAD + pidx].velocity.z) * WviscosityLaplacian(radiusSquared) / d_data[nidx][j].density;
										}
										colorFieldNormal.x += poly6Gradient.x / d_data[nidx][j].density;
										colorFieldNormal.y += poly6Gradient.y / d_data[nidx][j].density;
										colorFieldNormal.z += poly6Gradient.z / d_data[nidx][j].density;
										colorFieldLaplacian += Wpoly6Laplacian(radiusSquared) / d_data[nidx][j].density;
									}
									diffPosition.x = -d_data[idx][i * THREAD + pidx].position.x + d_data[nidx][j].position.x;
									diffPosition.y = -d_data[idx][i * THREAD + pidx].position.y + d_data[nidx][j].position.y;
									diffPosition.z = -d_data[idx][i * THREAD + pidx].position.z + d_data[nidx][j].position.z;
									if(iter > DRAGTIME && radiusSquared <= h * h && !d_data[nidx][j].exSPHflag){ //牵引粒子计算位置牵引力和速度牵引力
										f_attract.x += d_data[nidx][j].scaleFactor*diffPosition.x / sqrt(radiusSquared) * Wpoly6(radiusSquared);
										f_attract.y += d_data[nidx][j].scaleFactor*diffPosition.y / sqrt(radiusSquared) * Wpoly6(radiusSquared);
										f_attract.z += d_data[nidx][j].scaleFactor*diffPosition.z / sqrt(radiusSquared) * Wpoly6(radiusSquared);
										f_velocity.x += (d_data[nidx][j].velocity.x - d_data[idx][i * THREAD + pidx].velocity.x)*Wpoly6(radiusSquared);
										f_velocity.y += (d_data[nidx][j].velocity.y - d_data[idx][i * THREAD + pidx].velocity.y)*Wpoly6(radiusSquared);
										f_velocity.z += (d_data[nidx][j].velocity.z - d_data[idx][i * THREAD + pidx].velocity.z)*Wpoly6(radiusSquared);
									}
								}
							}
						}
					}
				}
				f_pressure.x *= -PARTICLE_MASS * d_data[idx][i * THREAD + pidx].density;
				f_pressure.y *= -PARTICLE_MASS * d_data[idx][i * THREAD + pidx].density;
				f_pressure.z *= -PARTICLE_MASS * d_data[idx][i * THREAD + pidx].density;
				f_viscosity.x = VISCOSITY * PARTICLE_MASS * f_viscosity.x;
				f_viscosity.y = VISCOSITY * PARTICLE_MASS * f_viscosity.y;
				f_viscosity.z = VISCOSITY * PARTICLE_MASS * f_viscosity.z;

				f_attract.x *= attractFactor;
				f_attract.y *= attractFactor;
				f_attract.z *= attractFactor;
				f_velocity.x *= velocityFactor;
				f_velocity.y *= velocityFactor;
				f_velocity.z *= velocityFactor;
				colorFieldNormal.x *= PARTICLE_MASS;
				colorFieldNormal.y *= PARTICLE_MASS;
				colorFieldNormal.z *= PARTICLE_MASS;
				/*d_data[idx][i * THREAD + pidx].normal.x = -1.0 * colorFieldNormal.x;
				d_data[idx][i * THREAD + pidx].normal.y = -1.0 * colorFieldNormal.y;
				d_data[idx][i * THREAD + pidx].normal.z = -1.0 * colorFieldNormal.z;*/
				colorFieldLaplacian *= PARTICLE_MASS;
				//int sqrtc;
				sqrtc = sqrt(colorFieldNormal.x * colorFieldNormal.x + colorFieldNormal.y * colorFieldNormal.y + colorFieldNormal.z * colorFieldNormal.z);
				// surface tension force
				if (sqrtc > SURFACE_THRESHOLD) {
					d_data[idx][i * THREAD + pidx].flag = true;
					f_surface.x = -SURFACE_TENSION * colorFieldNormal.x  * colorFieldLaplacian / sqrtc;
					f_surface.y = -SURFACE_TENSION * colorFieldNormal.y  * colorFieldLaplacian / sqrtc;
					f_surface.z = -SURFACE_TENSION * colorFieldNormal.z  * colorFieldLaplacian / sqrtc;
				}
				else {
					d_data[idx][i * THREAD + pidx].flag = false;
				}
				// ADD IN SPH FORCES
				d_data[idx][i * THREAD + pidx].acceleration.x = (f_pressure.x + f_viscosity.x + f_surface.x + f_gravity.x + f_attract.x + f_velocity.x) / d_data[idx][i * THREAD + pidx].density;
				d_data[idx][i * THREAD + pidx].acceleration.y = (f_pressure.y + f_viscosity.y + f_surface.y + f_gravity.y + f_attract.y + f_velocity.y) / d_data[idx][i * THREAD + pidx].density;
				d_data[idx][i * THREAD + pidx].acceleration.z = (f_pressure.z + f_viscosity.z + f_surface.z + f_gravity.z + f_attract.z + f_velocity.z) / d_data[idx][i * THREAD + pidx].density;
			}
		}
	}
}
__global__ void caculateCollision(particle** d_data, int* d_size, int xRes, int yRes, int zRes, wall* d_wall, int wallSize) {
	int idx = grid_index((int)blockIdx.x, (int)blockIdx.y, (int)blockIdx.z, xRes, yRes, zRes);
	// 访问grid(blockIdx.x, blockIdx.y, blockIdx.z)
	int pidx = (int)threadIdx.x;
	int i;
	if (idx < xRes*yRes*zRes) {
		for (i = 0; i * THREAD + pidx < d_size[idx]; ++i) {
			if (d_data[idx][i * THREAD + pidx].exSPHflag) {
				collisionForce_z(d_data[idx][i * THREAD + pidx], d_wall, wallSize);
			}
		}
	}
}

__global__ void updateGrid(particle** d_data, int* d_size, int xRes, int yRes, int zRes, particle** new_data, int* new_lock, int* new_size) {
	//将原始grid中的每个粒子放入新的grid中，新的particle in gird关系保存在new_data中
	int idx = grid_index((int)blockIdx.x, (int)blockIdx.y, (int)blockIdx.z, xRes, yRes, zRes);
	// 访问grid(blockIdx.x, blockIdx.y, blockIdx.z)
	int pidx = (int)threadIdx.x;
	// gird中的第（pidx + n*thread) 个粒子
	int i;
	if (idx < xRes*yRes*zRes) {
		for (i = 0; i * THREAD + pidx < d_size[idx]; ++i) {
			//计算该粒子所在的网格
			int x = (int)floor((d_data[idx][i * THREAD + pidx].position.x + BOXSIZE / 2.0) / h);
			int y = (int)floor((d_data[idx][i * THREAD + pidx].position.y + BOXSIZE / 2.0) / h);
			int z = (int)floor((d_data[idx][i * THREAD + pidx].position.z + BOXSIZE / 2.0) / h);
			if (x < 0) x = 0;
			else if (x >= xRes) x = xRes - 1;
			if (y < 0) y = 0;
			else if (y >= yRes) y = yRes - 1;
			if (z < 0) z = 0;
			else if (z >= zRes) z = zRes - 1;
			//将该粒子加入网格
			int newidx = grid_index(x, y, z, xRes, yRes, zRes);
			//如果该网格没有粒子正在加入即lock为false，则标记lock为true，开始加入粒子
			//if (atomicExch(&new_lock[newidx], 1)==0) {
			while (new_lock[newidx] != 0) {
				if(grid_index(x, y, z, xRes, yRes, zRes) == 5)
					printf("%d synchronized!!\n", (int)threadIdx.x);
			}
			atomicExch(&new_lock[newidx], 1);
			//d_data[idx][i * THREAD + pidx];
			//memcpy(&new_data[newidx][new_size[newidx]], &d_data[idx][i * THREAD + pidx], sizeof(particle));
			
			/*if (grid_index(x, y, z, xRes, yRes, zRes) == 5)
				printf("particles number in %d is %d\n", grid_index(x, y, z, xRes, yRes, zRes), new_size[newidx]);*/
			atomicAdd(&new_size[newidx], 1);
			atomicExch(&new_lock[newidx], 0);
		}
	}
}

void updataGridOnHost(particle** h_data, int* h_size, int xRes, int yRes, int zRes, particle** &new_data,  int* new_size) {
	for (int x = 0; x < xRes; ++x) {
		for (int y = 0; y < yRes; ++y) {
			for (int z = 0; z < zRes; ++z) {
				int idx = grid_index(x, y, z, xRes, yRes, zRes);
				for (int i = 0; i < h_size[idx]; ++i) {
					particle particle = h_data[idx][i];
					int newGridCellX = (int)floor((particle.position.x + BOXSIZEX / 2.0) / h);
					int newGridCellY = (int)floor((particle.position.y + BOXSIZEY / 2.0) / h);
					int newGridCellZ = (int)floor((particle.position.z + BOXSIZEZ / 2.0) / h);
					if (newGridCellX < 0)
						newGridCellX = 0;
					else if (newGridCellX >= xRes)
						newGridCellX = xRes - 1;
					if (newGridCellY < 0)
						newGridCellY = 0;
					else if (newGridCellY >= yRes)
						newGridCellY = yRes - 1;
					if (newGridCellZ < 0)
						newGridCellZ = 0;
					else if (newGridCellZ >= zRes)
						newGridCellZ = zRes - 1;
					int newidx = grid_index(newGridCellX, newGridCellY, newGridCellZ, xRes, yRes, zRes);
					/*new_data[newidx][new_size[newidx]].acceleration.x = particle.acceleration.x;
					new_data[newidx][new_size[newidx]].acceleration.y = particle.acceleration.y;
					new_data[newidx][new_size[newidx]].acceleration.z = particle.acceleration.z;
					new_data[newidx][new_size[newidx]].position.x = particle.position.x;
					new_data[newidx][new_size[newidx]].position.y = particle.position.y;
					new_data[newidx][new_size[newidx]].position.z = particle.position.z;
					new_data[newidx][new_size[newidx]].velocity.x = particle.velocity.x;
					new_data[newidx][new_size[newidx]].velocity.y = particle.velocity.y;
					new_data[newidx][new_size[newidx]].velocity.z = particle.velocity.z;
					new_data[newidx][new_size[newidx]].exSPHflag = particle.exSPHflag;
					new_data[newidx][new_size[newidx]].id = particle.id;*/
					//new_data[newidx][new_size[newidx]] = particle;
					//memcpy(&new_data[newidx][new_size[newidx]], &h_data[idx][i], sizeof(particle));
					new_size[newidx] += 1;
					//printf("(%d, %d, %d) newidx = %d size = %d\n ", newGridCellX, newGridCellY, newGridCellZ, newidx, new_size[newidx]);
				}
			}
		}
	}
}

__global__ void updatePosition(particle** d_data, int* d_size, int xRes, int yRes, int zRes, int iter) {
	int idx = grid_index((int)blockIdx.x, (int)blockIdx.y, (int)blockIdx.z, xRes, yRes, zRes);
	// 访问grid(blockIdx.x, blockIdx.y, blockIdx.z)
	int pidx = (int)threadIdx.x;
	int i;
	double dt = iter >= DRAGTIME ? 0.002 : 0.01;
	if (idx < xRes*yRes*zRes) {
		for (i = 0; i * THREAD + pidx < d_size[idx]; ++i) {
			if (d_data[idx][i * THREAD + pidx].exSPHflag) {
				d_data[idx][i * THREAD + pidx].position.x += d_data[idx][i * THREAD + pidx].velocity.x * dt + d_data[idx][i * THREAD + pidx].acceleration.x * dt * dt;
				d_data[idx][i * THREAD + pidx].position.y += d_data[idx][i * THREAD + pidx].velocity.y * dt + d_data[idx][i * THREAD + pidx].acceleration.y * dt * dt;
				d_data[idx][i * THREAD + pidx].position.z += d_data[idx][i * THREAD + pidx].velocity.z * dt + d_data[idx][i * THREAD + pidx].acceleration.z * dt * dt;
				d_data[idx][i * THREAD + pidx].velocity.x += d_data[idx][i * THREAD + pidx].acceleration.x * dt;
				d_data[idx][i * THREAD + pidx].velocity.y += d_data[idx][i * THREAD + pidx].acceleration.y * dt;
				d_data[idx][i * THREAD + pidx].velocity.z += d_data[idx][i * THREAD + pidx].acceleration.z * dt;
			}
		}
	}
}

void readPosition(FIELD_3D* field){
	for (int x = 0; x < xRes; x++) {
		for (int y = 0; y < yRes; y++) {
			for (int z = 0; z < zRes; z++) {
				vector<PARTICLE>& par = (*field)(x, y, z);
				h_size[grid_index(x, y, z, xRes, yRes, zRes)] = par.size();
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = (particle*)malloc(MAXINGRID * sizeof(particle));
				memset(h_data[grid_index(x, y, z, xRes, yRes, zRes)], 0, sizeof(particle) * MAXINGRID);
				//分配GPU空间存储粒子
				particle* d_temp;
				CHECK(hipMalloc((void**)&d_temp, MAXINGRID * sizeof(particle)));
				for (int p = 0; p < h_size[grid_index(x, y, z, xRes, yRes, zRes)]; ++p) {
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.x = par[p].position().x;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.y = par[p].position().y;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.z = par[p].position().z;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].id = par[p].id();
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].exSPHflag = true;
				}
				CHECK(hipMemcpy(d_temp, h_data[grid_index(x, y, z, xRes, yRes, zRes)], sizeof(particle) * MAXINGRID, hipMemcpyHostToDevice));
				free(h_data[grid_index(x, y, z, xRes, yRes, zRes)]);
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = d_temp;
			}//grid.x
		}//grid.y
	}//grid.z
	CHECK(hipMemcpy(d_size, h_size, sizeof(int) * xRes*yRes*zRes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_data, h_data, sizeof(particle*) * xRes*yRes*zRes, hipMemcpyHostToDevice));
}
extern "C" void gpu_run(FIELD_3D* field, vector<WALL> walls, int iter, bool loadwall) {
	//size用于记录每个gird中粒子的数量
	if (xRes != (*field).xRes() || yRes != (*field).yRes() || zRes != (*field).zRes()) {
		xRes = (*field).xRes();
		yRes = (*field).yRes();
		zRes = (*field).zRes();
		free(h_size);
		free(h_data);
		hipFree(d_size);
		hipFree(d_data);
		h_size = (int*)malloc(sizeof(int) * xRes * yRes * zRes);
		h_data = (particle**)malloc(sizeof(particle*) * xRes * yRes * zRes);
		CHECK(hipMalloc((void**)&d_data, sizeof(particle*) * xRes * yRes * zRes));
		CHECK(hipMalloc((void**)&d_size, sizeof(int)  * xRes * yRes * zRes));
		printf("cuda remalloc data!!\n");
		//readPosition(field);
	}
	for (int x = 0; x < xRes; x++) {
		for (int y = 0; y < yRes; y++) {
			for (int z = 0; z < zRes; z++) {
				vector<PARTICLE>& par = (*field)(x, y, z);
				h_size[grid_index(x, y, z, xRes, yRes, zRes)] = par.size();
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = (particle*)malloc(MAXINGRID * sizeof(particle));
				//memset(h_data[grid_index(x, y, z, xRes, yRes, zRes)], 0, sizeof(particle) * MAXINGRID);
				//分配GPU空间存储粒子
				particle* d_temp;
				CHECK(hipMalloc((void**)&d_temp, MAXINGRID * sizeof(particle)));
				for (int p = 0; p < h_size[grid_index(x, y, z, xRes, yRes, zRes)] && p < MAXINGRID; ++p) {
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.x = par[p].position().x;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.y = par[p].position().y;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.z = par[p].position().z;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].velocity.x = par[p].velocity().x;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].velocity.y = par[p].velocity().y;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].velocity.z = par[p].velocity().z;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].acceleration.x = par[p].acceleration().x;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].acceleration.y = par[p].acceleration().y;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].acceleration.z = par[p].acceleration().z;
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].id = par[p].id();
					h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].exSPHflag = par[p].exSPHflag();
				}
				CHECK(hipMemcpy(d_temp, h_data[grid_index(x, y, z, xRes, yRes, zRes)], sizeof(particle) * MAXINGRID, hipMemcpyHostToDevice));
				free(h_data[grid_index(x, y, z, xRes, yRes, zRes)]);
				//cout << "grid: " << x << "," << y << "," << z << endl;
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = NULL;
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = d_temp;
			}//grid.x
		}//grid.y
	}//grid.z
	CHECK(hipMemcpy(d_size, h_size, sizeof(int) * xRes*yRes*zRes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_data, h_data, sizeof(particle*) * xRes*yRes*zRes, hipMemcpyHostToDevice));
	//将wall数据存储到主机内存中
	if (loadwall) {
		h_wall = (wall*)malloc(sizeof(wall) * walls.size());
		CHECK(hipMalloc((void**)&d_wall, sizeof(wall) * walls.size()));
		for (int i = 0; i < walls.size(); ++i) {
			h_wall[i].normal.x = walls[i].normal().x;
			h_wall[i].normal.y = walls[i].normal().y;
			h_wall[i].normal.z = walls[i].normal().z;
			h_wall[i].point.x = walls[i].point().x;
			h_wall[i].point.y = walls[i].point().y;
			h_wall[i].point.z = walls[i].point().z;
		}
		CHECK(hipMemcpy(d_wall, h_wall, sizeof(wall) * walls.size(), hipMemcpyHostToDevice));
		free(h_wall);
	}
	//printf("class >>> struct  copy time: %lld ms\n", stop - start);
	dim3 block(xRes, yRes, zRes);
	dim3 thread(THREAD);
	caculateDensity << <block, thread >> > (d_data, d_size, xRes, yRes, zRes);
	caculateScaleFactor << <block, thread >> > (d_data, d_size, xRes, yRes, zRes);
	caculateAllForce << <block, thread >> > (d_data, d_size, xRes, yRes, zRes, iter);
	caculateCollision << <block, thread >> > (d_data, d_size, xRes, yRes, zRes, d_wall, walls.size());
	//计算新的位置
	updatePosition << <block, thread >> > (d_data, d_size, xRes, yRes, zRes, iter);
	//重新划分网格
	//  准备new_data, new_lock, new_size的内存空间
	CHECK(hipMemcpy(h_data, d_data, sizeof(particle*) * xRes*yRes*zRes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_size, d_size, sizeof(int) * xRes*yRes*zRes, hipMemcpyDeviceToHost));
	/*new_size = (int*)malloc(sizeof(int) * xRes * yRes * zRes);
	memset(new_size, 0, sizeof(int)  * xRes * yRes * zRes);
	particle** hnew_data = (particle**)malloc(xRes * yRes * zRes * sizeof(particle*));
	for (int x = 0; x < xRes; x++) {
		for (int y = 0; y < yRes; y++) {
			for (int z = 0; z < zRes; z++) {
				//分配GPU空间存储粒子
				particle* h_temp = (particle*)malloc(MAXINGRID * sizeof(particle));;
				particle* d2h_temp = (particle*)malloc(MAXINGRID * sizeof(particle));
				//CHECK(hipMalloc((void**)&d_temp, h_size[grid_index(x, y, z, xRes, yRes, zRes)] * sizeof(particle)));
				memset(h_temp, 0, MAXINGRID * sizeof(particle));
				memset(d2h_temp, 0, MAXINGRID * sizeof(particle));
				CHECK(hipMemcpy(d2h_temp, h_data[grid_index(x, y, z, xRes, yRes, zRes)], MAXINGRID * sizeof(particle), hipMemcpyDeviceToHost));
				hnew_data[grid_index(x, y, z, xRes, yRes, zRes)] = h_temp;
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = d2h_temp;
			}//grid.x
		}//grid.y
	}//grid.z*/
	//   On CPU
	/*updataGridOnHost(h_data, h_size, xRes, yRes, zRes, hnew_data, new_size);
	for (int i = 0; i < xRes * yRes * zRes; ++i) {
		free(h_data[i]);
	}
	free(h_data);
	h_data = hnew_data;
	free(h_size);
	h_size = new_size;*/
	//struct to class copy
	
	for (int x = 0; x < (*field).xRes(); x++) {
		for (int y = 0; y < (*field).yRes(); y++) {
			for (int z = 0; z < (*field).zRes(); z++) {
				vector<PARTICLE>& par = (*field)(x, y, z);
				vector<PARTICLE>::iterator itc = par.begin();
				h_size[grid_index(x, y, z, xRes, yRes, zRes)] = par.size();
				//分配CPU空间存储粒子
				particle* h_temp = (particle*)malloc(h_size[grid_index(x, y, z, xRes, yRes, zRes)] * sizeof(particle));;
				CHECK(hipMemcpy(h_temp, h_data[grid_index(x, y, z, xRes, yRes, zRes)], sizeof(particle) * h_size[grid_index(x, y, z, xRes, yRes, zRes)], hipMemcpyDeviceToHost));
				CHECK(hipFree(h_data[grid_index(x, y, z, xRes, yRes, zRes)]));
				h_data[grid_index(x, y, z, xRes, yRes, zRes)] = h_temp;
				for (int p = 0; p < h_size[grid_index(x, y, z, xRes, yRes, zRes)]; ++p) {
					if (abs(h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.x) > DX || abs(h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.z) > DZ) {
						itc = par.erase(itc);
						continue;
					}
					itc->acceleration().x = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].acceleration.x;
					itc->acceleration().y = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].acceleration.y;
					itc->acceleration().z = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].acceleration.z;
					itc->position().x = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.x;
					itc->position().y = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.y;
					itc->position().z = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].position.z;
					itc->velocity().x = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].velocity.x;
					itc->velocity().y = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].velocity.y;
					itc->velocity().z = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].velocity.z;
					itc->flag() = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].flag;
					itc->id() = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].id;
					itc->exSPHflag() = h_data[grid_index(x, y, z, xRes, yRes, zRes)][p].exSPHflag;
					++itc;
				}
			}//grid.x
		}//grid.y
	}//grid.z
	/*for (int x = 0; x < xRes; x++) {
		for (int y = 0; y < yRes; y++) {
			for (int z = 0; z < zRes; z++) {
				particle* d_temp;
				int idx = grid_index(x, y, z, xRes, yRes, zRes);
				CHECK(hipMalloc((void**)&d_temp, MAXINGRID * sizeof(particle)));
				CHECK(hipMemset(d_temp, 0, MAXINGRID * sizeof(particle)));
				CHECK(hipMemcpy(d_temp, h_data[idx], sizeof(particle) * MAXINGRID, hipMemcpyHostToDevice));
				free(h_data[idx]);
				h_data[idx] = d_temp;
			}//grid.x
		}//grid.y
	}//grid.z
	CHECK(hipMemcpy(d_size, h_size, sizeof(int) * xRes*yRes*zRes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_data, h_data, sizeof(particle*) * xRes*yRes*zRes, hipMemcpyHostToDevice));*/
	for (int i = 0; i < xRes*yRes*zRes; ++i)
		free(h_data[i]);
	return;
}

void remalloc(int _xRes, int _yRes, int _zRes) {
	if (_xRes != xRes || _yRes != yRes || _zRes != zRes) {
		CHECK(hipMalloc((void**)&d_data, sizeof(particle*) * xRes * yRes * zRes));
	}
}